#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>

using namespace std;

__global__ void ecualizacion(int *dev_img, int *dev_funcion){
    
    int x = threadIdx.x + blockIdx.x*blockDim.x;
  
    dev_img[x] = dev_funcion[dev_img[x]];

}

int main(){
    
    int w = 640;
    int h = 425;

    int img[h*w];

    // matriz de imagen y arreglo de funcion de ecualizacion

    ifstream archImg("/content/drive/My Drive/topicos en grafica/laboratorio 4/img.txt");
    if (!archImg.is_open()) cout<<"No se abrió el archivo\n";
  
    for(int i=0; i<h; i++)
        for(int j=0; j<w; j++)
            archImg >> img[i*w+j];
    archImg.close();

    ifstream archFuncion("/content/drive/My Drive/topicos en grafica/laboratorio 4/funcion.txt");
    if (!archFuncion.is_open()) cout<<"No se abrió el archivo\n";
  
    int funcion[256];
    for(int i=0; i<256; i++)
            archFuncion >> funcion[i];
    archFuncion.close();

    // proceso

    int *dev_img, *dev_funcion;

    hipMalloc((void **) &dev_img, sizeof(int)*w*h);
    hipMalloc((void **) &dev_funcion, sizeof(int)*256);

    hipMemcpy(dev_img, img, sizeof(int)*w*h, hipMemcpyHostToDevice);
    hipMemcpy(dev_funcion, funcion, sizeof(int)*256, hipMemcpyHostToDevice);

    ecualizacion<<<h,w>>>(dev_img, dev_funcion);

    hipMemcpy(img, dev_img, sizeof(int)*w*h, hipMemcpyDeviceToHost);

    ofstream arch("/content/drive/My Drive/topicos en grafica/laboratorio 4/imgEcualizada.txt");

    for(int i=0; i<h; i++){
      for (int j=0; j<w; j++)
          arch<<img[i*w+j]<<"\n";
    }

    arch.close();

    hipFree(dev_img);
    hipFree(dev_funcion);

    cout<<"ok."<<'\n';

    return 0;
}
