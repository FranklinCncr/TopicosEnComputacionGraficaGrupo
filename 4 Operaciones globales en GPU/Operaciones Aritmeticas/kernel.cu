#include "hip/hip_runtime.h"
#include <iostream>
#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <iostream>
#include <vector>
#include "BMP.h"

#include <stdio.h>
using namespace cv;
using namespace std;
// Suma por columnas de una matriz
__global__ void SumaMatrizKernel_1(int M, float* Md_1, float* Md_2, float* Md_3) {
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    Md_3[columna] = Md_2[columna] + Md_1[columna];
}

//suma por columnas
__global__ void SumaMatrizKernel_2(int N, float* Md_1, float* Md_2, float* Md_3,int chanales) {
    float Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;//blockIdx.x* blockDim.x + threadIdx.x;
    int posIni = blockIdx.x * blockDim.x + threadIdx.x; //columna * N;
    for (int k = 0; k < N; k+=1) {
        Md_3[chanales*(posIni*N + k)] = Md_2[chanales * (posIni * N + k)] + Md_1[chanales * (posIni * N + k)];
        Md_3[chanales * (posIni * N + k) +1] = Md_2[chanales * (posIni * N + k) +1] + Md_1[chanales * (posIni * N + k)+1];
        Md_3[chanales * (posIni * N + k) + 2] = Md_2[chanales * (posIni * N + k) + 2] + Md_1[chanales * (posIni * N + k)+2];
    }

}

// Suma por columnas de una matriz
__global__ void SumaColMatrizKernel_3(int M, float* Md_1, float* Md_2, float* Md_3) {
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Md_3[posIni + k] = Md_2[posIni + k] + Md_1[posIni + k];
    }
}

void SumaMatriz(int M, int N, float* Mh_1, float* Mh_2, float* Mr_3,int chanales) {
    int size = M * N * sizeof(float);
    float* Md_1;
    float* Md_2;
    float* Md_3;
    // Allocate en device
    hipMalloc(&Md_1, size);
    hipMalloc(&Md_2, size);
    hipMalloc(&Md_3, size);

    // Inicializo matrices en el device
    hipMemcpy(Md_1, Mh_1, size, hipMemcpyHostToDevice);
    hipMemcpy(Md_2, Mh_2, size, hipMemcpyHostToDevice);
    //SumaMatrizKernel_1 <<<1, M*N*3>>>(M, Md_1, Md_2, Md_3);
    //si hubiera tantos hilos suficientes
    
    SumaMatrizKernel_2 <<<1, M>>>(N, Md_1, Md_2, Md_3, chanales);
    /*int chunk = 2;
    SumaColMatrizKernel_3 << < N, M / chunk >> > (M, Md_1, Md_2, Md_3);*/
    hipMemcpy(Mr_3, Md_3, size, hipMemcpyDeviceToHost);
    cout << "Entra" << endl;
    int i = 0;
    while (i < M) {
        int j = 0;
        while (j < N) {
            cout << Mr_3[i*N+j] << " ";
            j += 1;
        }
        cout << endl;
        i += 1;
    }

    hipFree(Md_1);

}

int main() {
//profundidad 24

    BMP bmp_po("poesia.bmp");

    BMP bmp_po2("poesia_2.bmp");

    vector<uint8_t> datosMh_1 = bmp_po.get_data();
    vector<uint8_t> datosMh_2 = bmp_po2.get_data();

    cout << "total_datos" << datosMh_1.size() << endl;//150396 datos 
    int tamano_to = datosMh_1.size();
    int M = bmp_po.get_alto();
    int N = bmp_po.get_ancho();
    uint32_t chaneles = bmp_po.get_chaneles();
    float* Mh_1 = new float[tamano_to];//matriz host

    float* Mh_2 = new float[tamano_to];//matriz host

    float* Mr_3 = new float[tamano_to];//matriz host

    for (int i = 0; i < tamano_to; i += 1) {
        Mh_1[i] = datosMh_1[i];
        Mh_2[i] = datosMh_2[i];
        Mr_3[i] = 0;
    }


    /////////////////////SUMA SECUENCIA///////////////////////
    /*
   for (int i = 0; i < tamano_to; i += 3) {
       Mr_3[i] = datosMh_1[i] + datosMh_2[i];
   }*/
   

   /*
   for (int i = 0; i < M; i++) {
       for (int j = 0; j < N; j++) {
           Mr_3[chaneles*(i * N + j)+0] = datosMh_1[chaneles * (i * N + j)+0] + datosMh_2[chaneles * (i * N + j)+0];
           Mr_3[chaneles*(i * N + j) + 1] = datosMh_1[chaneles * (i * N + j) + 1] + datosMh_2[chaneles * (i * N + j) + 1];
           Mr_3[chaneles*(i * N + j) + 2] = datosMh_1[chaneles * (i * N + j) + 2] + datosMh_2[chaneles * (i * N + j) + 2];
       }
   }*/
   //va a saltando con la cantidad 
   /////////////////////////SUMA PARALELA////////////////////

    SumaMatriz(M, N, Mh_1, Mh_2, Mr_3, chaneles);

    vector<uint8_t> nuevo_data;
    for (int i = 0; i < tamano_to; i++) {
        nuevo_data.push_back(Mr_3[i]);
    }
    cout << "tamano de la imagen" << nuevo_data.size() << endl;

    bmp_po2.write_image_data("Salida_suma.bmp", nuevo_data);

    cout << "Erika animo t� puedes de verdad" << endl;

    return 0;
}