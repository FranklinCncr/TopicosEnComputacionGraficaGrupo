#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <math.h>
#include "iostream"
//Imagen de 10x10
#define SIZE	100
using namespace std;

__global__ void VectorAdd(int *a, int *r, int n, double gamma)
{
	int i=threadIdx.x;

	if(i<n)
		r[i] = (int)(255.0*pow((double)a[i]/255.0,1.0/gamma));
}

int main()
{
	int *a;
	int *r;
	double gamma=0.5;
	
	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&r, SIZE * sizeof(int));

	cout<<"Matriz de entrada"<<endl;

	for (int i = 1; i <= SIZE; ++i){
		a[i]=rand() % 255;
    cout<<a[i]<<"\t";
		if(i%10==0){
				cout<<endl;
		}
	}
		
	VectorAdd <<<1, SIZE>>> (a, r, SIZE, gamma);

  hipDeviceSynchronize();

	cout<<"Matriz de salida con Gamma = 0.5"<<endl;

	for (int i = 1; i <= SIZE; ++i){
    cout<<r[i]<<"\t";
		if(i%10==0){
				cout<<endl;
		}
	}

	hipFree(a);
	hipFree(r);

	return 0;
}