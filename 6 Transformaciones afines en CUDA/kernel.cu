#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <opencv2/highgui.hpp> // import no include errors
#include <opencv2/imgproc.hpp> // import no include errors 
#include <opencv2/core.hpp>       // import no include errors

#include <stdio.h>
#include <iostream>
#include <math.h>
using namespace std;
#define PI 3.14159265
__global__ void SumaMatrizKernel_1(int M, float* Md_1, float* Md_2, float* Md_3) {
    int columna = blockIdx.x * blockDim.x + threadIdx.x;

    Md_3[columna] = Md_2[columna] + Md_1[columna];
}

//suma por columnas
__global__ void SumaMatrizKernel_2(int N,int M, float* Md_1,  float* Md_3, int chanales) {
    float Pvalue = 0;
    int x = blockIdx.x;//*blockDim.x * 3;
    int y = threadIdx.x;//*3;
   
    int fila = blockIdx.x;
    int columna = threadIdx.x;
    int dx = 2;
    int dy = 1;

    int ix = 10;
    int iy = 10;

    int teta = 1;
    int arreglo[3][3] = { {1,0,dx},{0,1,dy},{0,0,1} };
    int pos[3][1] = { {fila},{columna},{1} };
    int salida[3][1] = { {0},{0},{0} };

    for (int i = 0;i < 3;i++) {
        for (int j = 0;j < 1;j++) {

            for (int k = 0;k < 3;k++) {
                salida[i][j] += arreglo[i][k] * pos[k][j];
            }
        }
    }
    int nuevo_fila = salida[0][0];
    int nuevo_columna = salida[1][0];
    //printf("(%d,%d) -> (%d,%d) \n", fila, columna, nuevo_fila, nuevo_columna);
    if (nuevo_fila < M & nuevo_columna < N) {
        Md_3[chanales * nuevo_fila*blockDim.x + nuevo_columna*chanales] = Md_1[chanales * x* blockDim.x + y*chanales];
        Md_3[chanales * nuevo_fila * blockDim.x + nuevo_columna * chanales+1] = Md_1[chanales * x* blockDim.x + y * chanales + 1];
        Md_3[chanales * nuevo_fila * blockDim.x + nuevo_columna * chanales+2] = Md_1[chanales * x* blockDim.x + y * chanales + 2];
    }
    else {
        int pos_px;
        int pos_py;
        if (x < M - 1) {
            pos_px = x;//M y N cambia /////falta arreglar
            pos_py = y - (M - 1);
        }
        else {
            pos_px = x-(N-1);//M y N cambia
            pos_py = y;
        }
        printf("(%d,%d) -> (%d,%d) \n", x, y, pos_px, pos_py);
        int vecinos[3] = {0,0,0};
        if (pos_px - 1 >= 0 & pos_py - 1 >= 0) {
            vecinos[0]+= Md_1[chanales * (pos_px-1) * blockDim.x + (pos_py - 1) * chanales + 0];
            vecinos[1] += Md_1[chanales * (pos_px - 1) * blockDim.x + (pos_py - 1) * chanales + 1];
            vecinos[2] += Md_1[chanales * (pos_px - 1) * blockDim.x + (pos_py - 1) * chanales + 2];
        }
        if (pos_px + 1 >= 0 & pos_py + 1 >= 0) {
            vecinos[0] += Md_1[chanales * (pos_px + 1) * blockDim.x + (pos_py + 1) * chanales ];
            vecinos[1] += Md_1[chanales * (pos_px + 1) * blockDim.x + (pos_py + 1) * chanales + 1];
            vecinos[2] += Md_1[chanales * (pos_px + 1) * blockDim.x + (pos_py + 1) * chanales + 2];
        }
        if (pos_px >= 0 & pos_py-1 >= 0) {
            vecinos[0] += Md_1[chanales * pos_px * blockDim.x + (pos_py - 1) * chanales + 0];
            vecinos[1] += Md_1[chanales * pos_px * blockDim.x + (pos_py - 1) * chanales + 1];
            vecinos[2] += Md_1[chanales * pos_px * blockDim.x + (pos_py - 1) * chanales + 2];
        }
        if (pos_px>= 0 & pos_py + 1 >= 0) {
            vecinos[0] += Md_1[chanales * pos_px * blockDim.x + (pos_py + 1) * chanales + 0];
            vecinos[1] += Md_1[chanales * pos_px * blockDim.x + (pos_py + 1) * chanales + 1 ];
            vecinos[2] += Md_1[chanales * pos_px * blockDim.x + (pos_py + 1) * chanales + 2];
        }
        if (pos_px < M & pos_py < N) {
           Md_3[chanales * pos_px * blockDim.x + pos_py * chanales] = vecinos[0];
            Md_3[chanales * pos_px * blockDim.x + pos_py * chanales + 1] = vecinos[1];
            Md_3[chanales * pos_px * blockDim.x + pos_py * chanales + 2] = vecinos[2];
        }
    }
}

// Suma por columnas de una matriz
__global__ void SumaColMatrizKernel_3(int M, float* Md_1, float* Md_2, float* Md_3) {
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Md_3[posIni + k] = Md_2[posIni + k] + Md_1[posIni + k];
    }
}

void SumaMatriz(int M, int N, float* Mh_1, float* Mr_3,  int chanales) {
    int size = M * N * 3 * sizeof(float);
    float* Md_1;
    float* Md_3;
    // Allocate en device
    hipMalloc(&Md_1, size);
    hipMalloc(&Md_3, size);

    // Inicializo matrices en el device
    hipMemcpy(Md_1, Mh_1, size, hipMemcpyHostToDevice);
    //SumaMatrizKernel_1 <<< 1,M*N*3 >>>(M, Md_1, Md_2, Md_3);
    //si hubiera tantos hilos suficientes
    /////////////oprimero es bloques y despues hilos
    /*int chunk = 2;
    SumaColMatrizKernel_3 << < N, M / chunk >> > (M, Md_1, Md_2, Md_3);*/
    SumaMatrizKernel_2 << < M, N >> > (N, M, Md_1, Md_3, chanales);
    hipMemcpy(Mr_3, Md_3, size, hipMemcpyDeviceToHost);
    cv::Mat imgSalidaFR(M, N, CV_32FC3, Mr_3);
    cv::imwrite("salida.jpg", imgSalidaFR);
    cv::Mat uno = cv::imread("salida.jpg", CV_32F);
    cv::imshow("Resultado", uno);

    hipFree(Md_1);

}

int main() {
    //profundidad 24


    cv::Mat A = cv::imread("d2.jpg", CV_32F);
    

    int M = A.rows;//alto
    int N = A.cols;
    
    int chaneles = A.channels();
    
    float* Mh_1 = new float[M * N * 3];//matriz host

    float* Mr_3 = new float[M * N * 3];//matriz host
    uchar * temp1=A.data;

    for (int i = 0; i < M * N * 3; i += 1) {
        Mh_1[i] = (float)temp1[i];

        Mr_3[i] = 0;
    }
    SumaMatriz(M, N, Mh_1, Mr_3, chaneles);
       
    cv::waitKey(0);
    return 0;
}