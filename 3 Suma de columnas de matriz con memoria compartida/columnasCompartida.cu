#include "hip/hip_runtime.h"

#include <bits/stdc++.h>

#define DIMBLOCKX 32

__global__ void SumaColMatrizKernel (int M, int N, float *Md, float *Nd){
  __shared__ float Nds[DIMBLOCKX];
  float Pvalue = 0;
  int columna = blockIdx.x * (N/gridDim.x) + threadIdx.x;
  int pasos = M/blockDim.x;
  int posIni = columna * M + threadIdx.x * pasos;

  for (int k=0; k<pasos; k++){
      Pvalue += Md[posIni + k];
  }

  atomicAdd(&Nd[columna],Pvalue);
  
  __syncthreads();

  if (threadIdx.x == 0){
      for (int i=1; i<blockDim.x; i++){
          Nds[0] += Nds[i];
      }
      atomicAdd(&Nd[blockIdx.x],Nds[0]);
  }
  
  
}  

void SumaColMatriz (int M, int N, float *Mh, float *Nh){
    
  int size = M*N*sizeof(float), size2=N*sizeof(float);
  float *Md, *Nd;

  hipMalloc(&Md, size);
  hipMalloc(&Nd, size2);

  hipMemcpy(Md, Mh, size,  hipMemcpyHostToDevice);
  hipMemset(Nd, 0, size2);

  int chunk = 32;
  dim3 tamGrid(N/chunk, 1);
  dim3 tamBlock(M/chunk, chunk, 1);
  SumaColMatrizKernel<<<tamGrid, tamBlock>>> (M, N, Md, Nd);

  hipMemcpy(Nh,Nd, size2, hipMemcpyDeviceToHost);

  hipFree(Md); hipFree(Nd);
}

int main(){

  int M = 1024;
  int N = 512;

  float *Mh = new float[M*N];//matriz host
  for (int i=0; i<M; i++)
    for (int j=0; j<N; j++)
      Mh[i*N+j] = 1;

  float *Nh = new float[N];//suma de columnas

  SumaColMatriz(M,N,Mh,Nh);

  for (int i=0; i<N; i++)
    std::cout<<Nh[i]<<" ";

  return 0;
}
